
#include <hip/hip_runtime.h>
﻿/*
	author: Krzysztof Sopyla (ksopyla@uwm.edu.pl)
	

*/


//computes two sparse matrix product in CRS format
//AVals - values for first matrix
//AIdx - indexes for first matrix
//APtrs - pointers to next vector
//BVals - values for second matrix
//BIdx - indexes for second matrix
//BPtrs - pointers to next vectors 
//result - result matrix
//ARows - number of rows in first matrix
//BCols - number of cols in second matrix
extern "C" __global__ void spmm_csr_naive(const float * AVals,
									   const int * AIdx, 
									   const int * APtrs,
									   const float * BVals,
									   const int * BIdx, 
									   const int * BPtrs,
									   float * result,
									   const int ARows,
									   const int BCols,
									   const int AElements,
									   const int BElements)
{

	const int row = blockIdx.y*blockDim.y+threadIdx.y;
	const int col = blockIdx.x*blockDim.x+threadIdx.x;
	
	if( !(row<ARows && col<BCols) )
	{
		return;
	}

	//possible optimization, cache this in shared memory
	int AStart = APtrs[row];
	int AEnd = APtrs[row+1];
	int curPosA = AStart;

	int BStart = BPtrs[col];
	int BEnd = BPtrs[col+1];
	int curPosB = BStart;

	int AcurIdx = AIdx[AStart];
	int BcurIdx = BIdx[BStart];
	

	float sum=0;

	while(curPosA<AEnd && curPosB<BEnd)
	{
		AcurIdx = AIdx[curPosA];
		BcurIdx = BIdx[curPosB];

		if(AcurIdx == BcurIdx)
		{
			sum+=AVals[curPosA]*BVals[curPosB];
			curPosA++;
			curPosB++;
		}else if( AcurIdx< BcurIdx)
		{
			curPosA++;
		}else
		{
			curPosB++;
		}

		

	}

	result[row*BCols+col] = sum;


}


//computes two sparse matrix product in CRS format, use shared memory to cache  
//one column vector in second matrix
//AVals - values for first matrix
//AIdx - indexes for first matrix
//APtrs - pointers to next vector
//BVals - values for second matrix
//BIdx - indexes for second matrix
//BPtrs - pointers to next vectors 
//result - result matrix
//ARows - number of rows in first matrix
//BCols - number of cols in second matrix
extern "C" __global__ void spmm_csr_naive_shared_one(const float * AVals,
									   const int * AIdx, 
									   const int * APtrs,
									   const float * BVals,
									   const int * BIdx, 
									   const int * BPtrs,
									   float * result,
									   const int ARows,
									   const int BCols,
									   const int AElements,
									   const int BElements)
{
	//max size = 4081
	__shared__ int svIdx[121];
	__shared__ float svVals[121];

	//barier[0]=BStart
	//barier[1]=BEnd
	__shared__ int barier[2];
	
	const int row = blockIdx.y*blockDim.y+threadIdx.y;
	const int col = blockIdx.x*blockDim.x+threadIdx.x;
	
	if( !(row<ARows && col<BCols) )
	{
		return;
	}

	//int BStart = BPtrs[col];
	if(threadIdx.y<2){
		barier[threadIdx.y]=BPtrs[col+threadIdx.y]	;
	}
	//????
	__syncthreads();
	int curPosB = barier[0];
	int diff=barier[1]-barier[0];
	
	//int curPosB = BPtrs[col];
	//int diff = BPtrs[col+1] - curPosB;

	int BcurIdx;

	for(int th=threadIdx.y; th<diff;th+=blockDim.y)
	{
		svVals[th]= BVals[curPosB+th];
		svIdx[th]=BIdx[curPosB+th];
	}
	__syncthreads();

	int curPosA = APtrs[row];
	int AEnd = APtrs[row+1];
	int AcurIdx;
	float sum=0;
	//now B column is in shared mem, so it starts from 0
	curPosB=0;
	
	while(curPosA<AEnd && curPosB<diff)
	{
		AcurIdx = AIdx[curPosA];
		BcurIdx = svIdx[curPosB];

		if(AcurIdx == BcurIdx)
		{
			sum+=AVals[curPosA]*svVals[curPosB];
			curPosA++;
			curPosB++;
		}else if( AcurIdx< BcurIdx)
		{
			curPosA++;
		}else
		{
			curPosB++;
		}
	}
	__syncthreads();
	result[row*BCols+col] = sum;
	//column major order
	//result[row+ARows*col] = sum;
}
