#include "hip/hip_runtime.h"
﻿

#define BLOCKSIZE 16

dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
dim3 dimGrid(BCols/dimBlock.x, ARows/dimBlock.y);

//computes two sparse matrix product in CRS format
//AVals - values for first matrix
//AIdx - indexes for first matrix
//APtrs - pointers to next vector
//BVals - values for second matrix
//BIdx - indexes for second matrix
//BPtrs - pointers to next vectors 
//result - result matrix
//ARows - number of rows in first matrix
//BCols - number of cols in second matrix
extern "C" __global__ void spmm_csr_scalar(const float * AVals,
									   const int * AIdx, 
									   const int * APtrs,
									   const float * BVals,
									   const int * BIdx, 
									   const int * BPtrs,
									   float * result,
									   const int ARows,
									   const int BCols)
{

	const int row = blockIdx.y*blockDim.y+threadIdx.y;
	const int col = blockIdx.x*blockDim.x+threadIdx.x;
	
	if( !(row<ARows && col<BCols) )
	{
		return;
	}

	//possible optimization, cache this in shared memory
	int AStart = APtrs[row];
	int AEnd = APtrs[row+1];

	int BStart = BPtrs[col];
	int BEnd = BPtrs[col+1];

	int AcurIdx = AIdx[AStart];
	int BcurIdx = BIdx[BStart];

	float sum=0;
	

	while(AcurIdx<AEnd && BcurIdx<BEnd)
	{
		if(AIdx[AcurIdx] == BIdx[BcurIdx])
		{
			sum+=AVals[AcurIdx]*BVals[BcurIdx];
			AcurIdx++;
			BcurIdx++;
		}else if( AIdx[AcurIdx]< BIdx[BcurIdx])
		{
			AcurIdx++;
		}else
		{
			BcurIdx++;
		}

	}

	result[row*ARows+col] = sum;


}
