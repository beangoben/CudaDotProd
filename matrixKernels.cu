
#include <hip/hip_runtime.h>
﻿/*
	author: Krzysztof Sopyla (ksopyla@uwm.edu.pl)
	

*/


//computes two sparse matrix product in CRS format
//AVals - values for first matrix
//AIdx - indexes for first matrix
//APtrs - pointers to next vector
//BVals - values for second matrix
//BIdx - indexes for second matrix
//BPtrs - pointers to next vectors 
//result - result matrix
//ARows - number of rows in first matrix
//BCols - number of cols in second matrix
extern "C" __global__ void spmm_csr_naive(const float * AVals,
									   const int * AIdx, 
									   const int * APtrs,
									   const float * BVals,
									   const int * BIdx, 
									   const int * BPtrs,
									   float * result,
									   const int ARows,
									   const int BCols,
									   const int AElements,
									   const int BElements)
{

	const int row = blockIdx.y*blockDim.y+threadIdx.y;
	const int col = blockIdx.x*blockDim.x+threadIdx.x;
	
	if( !(row<ARows && col<BCols) )
	{
		return;
	}

	//possible optimization, cache this in shared memory
	int AStart = APtrs[row];
	int AEnd = APtrs[row+1];
	int curPosA = AStart;

	int BStart = BPtrs[col];
	int BEnd = BPtrs[col+1];
	int curPosB = BStart;

	int AcurIdx = AIdx[AStart];
	int BcurIdx = BIdx[BStart];
	

	float sum=0;

	while(curPosA<AEnd && curPosB<BEnd)
	{
		AcurIdx = AIdx[curPosA];
		BcurIdx = BIdx[curPosB];

		if(AcurIdx == BcurIdx)
		{
			sum+=AVals[curPosA]*BVals[curPosB];
			curPosA++;
			curPosB++;
		}else if( AcurIdx< BcurIdx)
		{
			curPosA++;
		}else
		{
			curPosB++;
		}

		

	}

	result[row*BCols+col] = sum;


}


//computes two sparse matrix product in CRS format, use shared memory to cache  
//one column vector in second matrix
//AVals - values for first matrix
//AIdx - indexes for first matrix
//APtrs - pointers to next vector
//BVals - values for second matrix
//BIdx - indexes for second matrix
//BPtrs - pointers to next vectors 
//result - result matrix
//ARows - number of rows in first matrix
//BCols - number of cols in second matrix
extern "C" __global__ void spmm_csr_naive_shared_one(const float * AVals,
									   const int * AIdx, 
									   const int * APtrs,
									   const float * BVals,
									   const int * BIdx, 
									   const int * BPtrs,
									   float * result,
									   const int ARows,
									   const int BCols,
									   const int AElements,
									   const int BElements)
{
	//max size = 4081
	__shared__ float svIdx[1000];
	__shared__ float svVals[1000];

	
	const int row = blockIdx.y*blockDim.y+threadIdx.y;
	const int col = blockIdx.x*blockDim.x+threadIdx.x;
	
	//int BStart = BPtrs[col];
	int BEnd = BPtrs[col+1];
	int curPosB = BPtrs[col];
	
	int diff=BEnd-curPosB;
	
	int BcurIdx;

	for(int th=threadIdx.y; th<(BEnd-curPosB);th+=blockDim.y)
	{
		svVals[th]= BVals[curPosB+th];
		svIdx[th]=BIdx[curPosB+th];
	}
	__syncthreads();

	if( !(row<ARows && col<BCols) )
	{
		return;
	}

	//possible optimization, cache this in shared memory
	//int AStart = APtrs[row];
	int AEnd = APtrs[row+1];
	int curPosA = APtrs[row];
	int AcurIdx;

	float sum=0;

	//now B column is in shared mem, so it starts from 0
	curPosB=0;
	BEnd=diff;

	while(curPosA<AEnd && curPosB<BEnd)
	{
		AcurIdx = AIdx[curPosA];
		BcurIdx = svIdx[curPosB];

		if(AcurIdx == BcurIdx)
		{
			sum+=AVals[curPosA]*svVals[curPosB];
			curPosA++;
			curPosB++;
		}else if( AcurIdx< BcurIdx)
		{
			curPosA++;
		}else
		{
			curPosB++;
		}

	}

	result[row*BCols+col] = sum;


}
