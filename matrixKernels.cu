
#include <hip/hip_runtime.h>
﻿

#define BLOCKSIZE 16



//computes two sparse matrix product in CRS format
//AVals - values for first matrix
//AIdx - indexes for first matrix
//APtrs - pointers to next vector
//BVals - values for second matrix
//BIdx - indexes for second matrix
//BPtrs - pointers to next vectors 
//result - result matrix
//ARows - number of rows in first matrix
//BCols - number of cols in second matrix
extern "C" __global__ void spmm_csr_naive(const float * AVals,
									   const int * AIdx, 
									   const int * APtrs,
									   const float * BVals,
									   const int * BIdx, 
									   const int * BPtrs,
									   float * result,
									   const int ARows,
									   const int BCols,
									   const int AElements,
									   const int BElements)
{

	const int row = blockIdx.y*blockDim.y+threadIdx.y;
	const int col = blockIdx.x*blockDim.x+threadIdx.x;
	
	if( !(row<ARows && col<BCols) )
	{
		return;
	}

	//possible optimization, cache this in shared memory
	int AStart = APtrs[row];
	int AEnd = APtrs[row+1];
	int curPosA = AStart;

	int BStart = BPtrs[col];
	int BEnd = BPtrs[col+1];
	int curPosB = BStart;

	int AcurIdx = AIdx[AStart];
	int BcurIdx = BIdx[BStart];
	

	float sum=0;

	while(curPosA<AEnd && curPosB<BEnd)
	{
		AcurIdx = AIdx[curPosA];
		BcurIdx = BIdx[curPosB];

		if(AcurIdx == BcurIdx)
		{
			sum+=AVals[curPosA]*BVals[curPosB];
			curPosA++;
			curPosB++;
		}else if( AcurIdx< BcurIdx)
		{
			curPosA++;
		}else
		{
			curPosB++;
		}

		/*
		if(curPosA<AElements)
			AcurIdx = AIdx[curPosA];
		
		if(curPosB<BElements)
			BcurIdx = BIdx[curPosB];
			*/

	}

	result[row*BCols+col] = sum;


}
