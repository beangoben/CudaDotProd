#include "hip/hip_runtime.h"
﻿


#define VEC_SIZE  200

typedef struct  strSparseVec
{
	int size;
	float* values;
	int* indices;
} SparseVec;



//simple kernel for adding two vectors
extern "C" __global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}


//kernel only for testing how to pass pointer to STRUCT from .net 
extern "C" __global__ void StructPass(const SparseVec* vec,float* out,int N)
{

    int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i < N){

		/*out[i]+=vec[i].size+N;

		int idx = vec[i].indices[0];
		out[i]+=idx;*/

		 for(int k=0;k<vec[i].size;k++)
		 {
			 //int idx = vec[i].indices[k];
			 out[i] += vec[i].values[k];
		 }
	}
}


//Kernel with pointer to Struct which contains sparse vector values and indices
//not efficient
extern "C" __global__ void DotProd(const SparseVec* vec,float* out,int mainIdx, int N)
{
	__shared__ SparseVec  vec1;
	
	if(threadIdx.x==0){
		vec1= vec[mainIdx];
	}

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= N){
		return;
	}
	SparseVec vec2 = vec[i];

	__syncthreads();

	int curr1=0;
	int curr2=0;

	int idx1=0;
	int idx2=0;
	float result =0;
	while (curr1 < vec1.size && curr2 < vec2.size)
	{
		idx1 = vec1.indices[curr1];
		idx2 = vec2.indices[curr2];

		if (idx1 == idx2)
		{
			result += vec1.values[curr1] * vec2.values[curr2];
			curr1++; curr2++;
		}
		else if (idx1 < idx2)
		{
			curr1++;
		}
		else
		{
			curr2++;
		}
	}

	out[i]=result;

}

//
extern "C" __global__ void DotProd2(const SparseVec* vec,float* out,int mainIdx, int N)
{
	__shared__ SparseVec  vec1;
	__shared__ float vec1Values[VEC_SIZE];
	__shared__ int vec1Indices[VEC_SIZE];
	__shared__ int vec1Size;

	if(threadIdx.x==0){
		vec1= vec[mainIdx];
		vec1Size =vec1.size;

		for(int k=0;k<vec1Size;k++)
		{
			vec1Values[k]=vec1.values[k];

			vec1Indices[k]=vec1.indices[k];

		}

	}

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= N){
		return;
	}
	SparseVec vec2 = vec[i];

	__syncthreads();

	int curr1=0;
	int curr2=0;

	int idx1=0;
	int idx2=0;
	float result =0;
	while (curr1 < vec1Size && curr2 < vec2.size)
	{
		idx1 = vec1Indices[curr1];
		idx2 = vec2.indices[curr2];

		if (idx1 == idx2)
		{
			result += vec1Values[curr1] * vec2.values[curr2];
			curr1++; curr2++;
		}
		else if (idx1 < idx2)
		{
			curr1++;
		}
		else
		{
			curr2++;
		}
	}

	out[i]=result;

}


//bad IDEA !!
//extern "C" __global__ void DotProd3(const SparseVec* vec,float* out,int mainIdx, int N)
//{
//	__shared__ SparseVec  vec1;
//	__shared__ float vec1Values[VEC_SIZE];
//	__shared__ int vec1Indices[VEC_SIZE];
//	__shared__ int vec1Size;
//
//	__shared__ float vecVals[BLOCK_SIZE][VEC_SIZE];
//	__shared__ int vecIdx[BLOCK_SIZE][VEC_SIZE];
//	__shared__ int vecSizes[BLOCK_SIZE];
//
//	if(threadIdx.x==0){
//		vec1= vec[mainIdx];
//		vec1Size =vec1.size;
//
//		for(int k=0;k<vec1Size;k++)
//		{
//			vec1Values[k]=vec1.values[k];
//
//			vec1Indices[k]=vec1.indices[k];
//		}
//
//	}
//
//	int i = blockDim.x * blockIdx.x + threadIdx.x;
//	if (i >= N){
//		return;
//	}
//	for(int j=0;j<BLOCK_SIZE;j++)
//	{
//		int ii = blockDim.x * blockIdx.x + j;
//		if(ii<N){
//		SparseVec vector= vec[ii];
//
//		if(threadIdx.x<vector.size)
//		{
//			vecVals[j][threadIdx.x]=vector.values[threadIdx.x];
//			vecIdx[j][threadIdx.x]=vector.indices[threadIdx.x];
//			vecSizes[j]=vector.size;
//		}
//		}
//	}
//	
//
//	__syncthreads();
//
//	int curr1=0;
//	int curr2=0;
//
//	int idx1=0;
//	int idx2=0;
//	float result =0;
//	while (curr1 < vec1Size && curr2 < vecSizes[i])
//	{
//		idx1 = vec1Indices[curr1];
//		idx2 = vecIdx[i][curr2];
//
//		if (idx1 == idx2)
//		{
//			result += vec1Values[curr1] * vecVals[i][curr2];
//			curr1++; curr2++;
//		}
//		else if (idx1 < idx2)
//		{
//			curr1++;
//		}
//		else
//		{
//			curr2++;
//		}
//	}
//
//	out[i]=result;
//
//}



//sparse matrix vector multiplication matrix in ELLPack format
extern "C" __global__ void DotProdEllPack(const float* vals,
										  const int* idx,
										  const float* mainVec,
										  float* out,
										  int maxRowSize,
										  int N)
{


	int row = blockDim.x*blockIdx.x+threadIdx.x;

	if(row<N){
		float dot=0;
		for(int i=0;i<maxRowSize;i++)
		{
			int col = idx[N*i+row];
			float val = vals[N*i+row];

			if(val!=0)
			{
				dot+=val*mainVec[col];
			}
		}

		out[row]+=dot;
	}
}

texture<float,1,hipReadModeElementType> texRef;

//sparse matrix vector multiplication matrix in ELLPack format, vector in texture "cache"
//better preformace than previous DotProdEllPack kernel
extern "C" __global__ void DotProdEllPackCached(const float* vals,
										  const int* idx,
										  float* out,
										  int maxRowSize,
										  int N)
{


	int row = blockDim.x*blockIdx.x+threadIdx.x;

	if(row<N){
		float dot=0;
		for(int i=0;i<maxRowSize;i++)
		{
			int col = idx[N*i+row];
			float val = vals[N*i+row];

			if(val!=0)
			{
				dot+=val* tex1D(texRef,col); 
			}
		}

		out[row]+=dot;
	}
}


//simple multiplication one element from matrix times one element from vector,
//only for testing multiplciation speed - its realy fast!
extern "C" __global__ void SegmentedMulCached(const float* vals,
										  const int* idx,
										  float* out,
										  int N)
{

	int row = blockDim.x*blockIdx.x+threadIdx.x;

	if(row<N){
		
		out[row]=vals[row]*tex1D(texRef,idx[row]);
		
	}
}


//sparse matrix-vector multiplication based one experiments from above kernel, 
//quite fast
extern "C" __global__ void DotProdSegmentedCached(const float* vals,
										  const int* idx,
										  const int* vecLenght,
										  float* temp,
										  float* dotArr,
										  int numRows,
										  int numElements)
{

	int row = blockDim.x*blockIdx.x+threadIdx.x;

	if(row<numElements){

		temp[row]=vals[row]*tex1D(texRef,idx[row]);

		__syncthreads();


		if(row<numRows){
			float dot=0;

			for(int k=vecLenght[row];k<vecLenght[row+1];k++)
			{
				dot+=temp[k];
			}

			dotArr[row]=dot;
		}
	}
}



#define BLOCK_SIZE 256

#define WARP_SIZE 32
extern "C" __global__ void spmv_csr_vector_kernel(const float * Ax,
									   const int * Aj, 
									   const int * Ap, 
									   float * y,
									   const int num_rows,
									   int numElements)
{
    __shared__ float sdata[BLOCK_SIZE + 16];                          // padded to avoid reduction ifs
    __shared__ int ptrs[BLOCK_SIZE/WARP_SIZE][2];
    
    const int thread_id   = BLOCK_SIZE * blockIdx.x + threadIdx.x;  // global thread index
    const int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
    const int warp_id     = thread_id   / WARP_SIZE;                // global warp index
    const int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
    const int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // total number of active warps

    for(int row = warp_id; row < num_rows; row += num_warps){
        // use two threads to fetch Ap[row] and Ap[row+1]
        // this is considerably faster than the straightforward version
        if(thread_lane < 2)
            ptrs[warp_lane][thread_lane] = Ap[row + thread_lane];
        const int row_start = ptrs[warp_lane][0];                   //same as: row_start = Ap[row];
        const int row_end   = ptrs[warp_lane][1];                   //same as: row_end   = Ap[row+1];

        // compute local sum
        float sum = 0;
        for(int jj = row_start + thread_lane; jj < row_end; jj += WARP_SIZE)
            sum += Ax[jj] * tex1D(texRef,Aj[jj]);

        // reduce local sums to row sum (ASSUME: warpsize 32)
        sdata[threadIdx.x] = sum;
        sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x + 16]; __syncthreads(); 
        sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  8]; __syncthreads();
        sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  4]; __syncthreads();
        sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  2]; __syncthreads();
        sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  1]; __syncthreads();
       


        // first thread writes warp result
        if (thread_lane == 0)
            y[row] += sdata[threadIdx.x];
    }
}
